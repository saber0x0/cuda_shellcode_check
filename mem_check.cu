#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
//#include<yara.h>
/*
��CUDA�У�����ֱ�ӱ���GPU�ڴ棬��ΪGPU�ڴ��������CUDA����ʱ����ģ�����GPU�ڴ��ַ��������CPU����˵�ǲ�͸���ġ�
��ʼ��GPU�ڴ棬Ȼ�󽫳�ʼ�������ݸ��Ƶ�ͳһ�ڴ������ٴ�ͳһ�ڴ渴�Ƶ������ڴ�, ��������ֱ�Ӷ��ǿ��Ե�, ��װһ��mallocҲ�����������͸��
//ͳһ�ڴ棨Unified Memory��/�����ڴ�ӳ�䣨Host Memory Mapping��/�㿽���ڴ棨Zero-copy Memory��/ֱ�������ڴ���ʣ�Direct Host Memory Access��/ҳ�����ڴ棨Page-locked Memory��
*/

// CUDA�ں�copyDataToUnifiedMemory
__global__ void
copyDataToUnifiedMemory(const char* src, char* dst, size_t size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = src[idx];
    }
}

__global__ void 
initializeMemory(int* dst, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        dst[idx] = idx;
    }
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

typedef struct MemoryBlock {
    void* ptr;
    size_t size;
    hipMemoryType memoryType;
    struct MemoryBlock* next;
} MemoryBlock;

//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

MemoryBlock* head = NULL;

//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

void* 
bcMalloc(size_t size, hipMemoryType memoryType) {
    void* ptr;
    hipError_t result = hipMalloc(&ptr, size);
    if (result == hipSuccess) {
        MemoryBlock* block = (MemoryBlock*)malloc(sizeof(MemoryBlock));
        block->ptr = ptr;
        block->size = size;
        block->memoryType = memoryType;
        block->next = head;
        head = block;
        return ptr;
    }
    else {
        fprintf(stderr, "CUDA malloc failed: %s\n", hipGetErrorString(result));
        return NULL;
    }
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

void 
bcFree(void* ptr) {
    MemoryBlock* current = head;
    MemoryBlock* previous = NULL;
    while (current != NULL) {
        if (current->ptr == ptr) {
            if (previous == NULL) {
                head = current->next;
            }
            else {
                previous->next = current->next;
            }
            hipFree(ptr);
            free(current);
            return;
        }
        previous = current;
        current = current->next;
    }
    fprintf(stderr, "Attempted to free untracked memory pointer.\n");
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

size_t 
getAllocatedMemory() {
    MemoryBlock* current = head;
    size_t total = 0;
    while (current != NULL) {
        total += current->size;
        current = current->next;
        // printf("���ַ:%d\n���С:%d\n��һ���С:%d\n", current->ptr, current->size, current->next);
        // �豸�ڴ�/�����ڴ�/ͳһ�ڴ�/�����ڴ�/�����ڴ�/�����ڴ�
    }
    return total;
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

void printMemoryType(void* ptr) {
    hipPointerAttribute_t attributes;
    hipError_t result = hipPointerGetAttributes(&attributes, ptr);
    if (result != hipSuccess) {
        fprintf(stderr, "hipPointerGetAttributes failed: %s\n", hipGetErrorString(result));
        return;
    }
    switch (attributes.type) {
    case hipMemoryTypeHost:
        printf("�ڴ�����: �����ڴ�\n");
        break;
    case hipMemoryTypeDevice:
        printf("�ڴ�����: �豸�ڴ�\n");
        break;
    case hipMemoryTypeManaged:
        printf("�ڴ�����: ͳһ�ڴ�\n");
        break;
    case cudaMemoryTypeUnregistered:
    default:
        printf("δ֪��δע����ڴ�����\n");
        break;
    }
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

void printAllocatedBlocks() {
    MemoryBlock* current = head;
    while (current != NULL) {
        printf("Memory Block:\n");
        printf("  Pointer: %p\n", current->ptr);
        printf("  Size: %zu\n", current->size);

        // ��ӡ�ڴ�����
        switch (current->memoryType) {
        case hipMemoryTypeHost:
            printf("  Memory Type: Host\n");
            break;
        case hipMemoryTypeDevice:
            printf("  Memory Type: Device\n");
            break;
        case hipMemoryTypeManaged:
            printf("  Memory Type: Managed (Unified)\n");
            break;
        case cudaMemoryTypeUnregistered:
            printf("  Memory Type: Unregistered\n");
            break;
        default:
            printf("  Memory Type: Unknown\n");
            break;
        }
        current = current->next;
    }
}
//------------------------------------------------------------------------------------------------------------------------------------------------------------------------//

int
main() {
    // ת�����ڴ��С1KB
    const size_t size = 1024;
    size_t totol;
    // �豸ָ��
    char* d_src, * d_dst, * h_dst, * h_dst2, * h_dst3;
    // ������һ��ͳһ�ڴ�����d_dst, ������ͳһ�ڴ�
    hipMallocManaged(&d_dst, size);
    // ��������һ���豸�ڴ�d_src
    // �������d_src����֪�ģ�ָ������Ҫת����GPU�ڴ�����
    hipMalloc(&d_src, size);
    printMemoryType((void*)d_src);


    // �����߳̿�������С
    int blockSize = 256;
    int numBlocks = (size + blockSize - 1) / blockSize;
    // ��ʼ��
    // initializeMemory << <numBlocks, blockSize >> > ((int*)d_src, size);
    // ��ʼ��ʾ��
    hipMemset(d_src, 'A', size);
    printMemoryType((void*)d_dst);

    // ���ݸ��Ƶ�ͳһ�ڴ�
    copyDataToUnifiedMemory << <numBlocks, blockSize >> > (d_src, d_dst, size);

    // �ȴ�GPU���
    hipDeviceSynchronize();

    // �����ں˺�����ʼ���豸�ڴ�

    // ��ͳһ�ڴ�����d_dst�����ݸ��Ƶ������ڴ�h_dst
    h_dst = (char*)malloc(size);
    hipMemcpy(h_dst, d_dst, size, hipMemcpyDeviceToHost);
    printMemoryType((void*)h_dst);
    printf("shellcode1:\n%s\n", h_dst);

    hipFree(d_src);
    hipFree(d_dst);
    free(h_dst);

    // ��װ��hipMalloc��hipFree
    void* n_malloc = bcMalloc(size, hipMemoryTypeDevice);
    hipMemset(n_malloc, 'B', size);
    h_dst2 = (char*)malloc(size);
    hipMemcpy(h_dst2, n_malloc, size, hipMemcpyDeviceToHost);
    printMemoryType(n_malloc);
    printf("shellcode2:\n%s\n", h_dst2);
    //totol = getAllocatedMemory();
    //printf("������ڴ��С:\n%d\n", (int)totol);
    h_dst3 = (char*)malloc(size);
    hipMemcpy(h_dst3, (char*)0x703000000, size, hipMemcpyDeviceToHost);
    printf("shellcode22:\n%s\n", h_dst3);
    printAllocatedBlocks();
    bcFree(n_malloc);
    return 0;
}